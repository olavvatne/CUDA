
#include "hip/hip_runtime.h"


#include <stdio.h>
#define SIZE 1024

// Global functions - kernels. Device code. Run on GPU. Code that run on CPU is host code
__global__
void VectorAdd(int *a, int *b, int *c, int n) {
	int i = threadIdx.x; // A readonly variable

	// if more threads than elements
	if (i < n) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
		
	}

	VectorAdd <<<1, SIZE>>> (a, b, c, SIZE);

	hipDeviceSynchronize();
	for (int i = 0; i < 10; ++i) {
		printf("c[%d] = %d\n", i, c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
