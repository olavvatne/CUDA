
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

		for (int i = index; i < n; i += stride) {
			y[i] = x[i] + y[i];
		}

}

int main(void) {
	int N = 1 << 20;
	float *x, *y;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;

	printf("Runs with %d blocks \n", numBlocks);
	// Run kernel on 1M elements on the GPU
	add <<<numBlocks, blockSize>>>(N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	for (int i = 0; i < 10; i++) {
		printf("[%d] is %f \n", i, y[i]);
	}

	hipFree(x);
	hipFree(y);

	return 0;
}